#include "hip/hip_runtime.h"
/*
Neste exercício, você deve implementar uma soma de prefixos em CUDA.

Você deve completar o código em três pontos:

## SOMA DE PREFIXOS EM CPU ##

Implemente a soma de prefixos em CPU e execute sobre o vetor original.
Utilize a saída do algoritmo síncrono (em CPU) para verificar a
corretude da saída do algoritmo paralelo (em GPU)

## SOMA DE PREFIXOS EM GPU ##

Utilize seus conhecimentos em CUDA para implementar uma estratégia
de indexação de threads para computar a soma de prefixos em paralelo.

## VERIFICAÇÃO ##

Implemente algum tipo de verificação. Sugestão: confira
se a saída do algoritmo síncrono (CPU) bate com a saída do
algoritmo paralelo (GPU)
*/
#include <cstdlib>
#include <iostream>
#include <vector>
#include <algorithm>
#include <cassert>
#include <numeric>

using std::generate;
using std::cout;
using std::vector;

#define TAMANHO_MEM_COMPARTILHADA 256

__global__ void somaPrefixos(int *v, int *v_somas) {
	// OBSERVAÇÃO: Não é necessário implementar
	// utilizando memória compartilhada, apesar de ser
	// a alternativa ótima (para que threads de um mesmo bloco)
	// não precisem acessar a memória global reiteradas vezes.

	// Cálculo do ID da thread
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	// ## SOMA DE PREFIXOS EM GPU ##
	// Implemente aqui a soma de prefixos paralela
}

int main() {
	// Tamanho do array: 8
	int N = 1 << 3;
	size_t bytes = N * sizeof(int);

	// Vetores (arrays) na maquina host: vetor original e
    // vetor reduzido.
	vector<int> host_arr(N);
	vector<int> host_somas_prefixos(N);

    // Inicializa o vetor , i. e., aloca memória na maquina host
    generate(begin(host_arr), end(host_arr), [](){ return rand() % 10; });

	// Aloca memória no dispositivo (device)
	int *device_arr, *device_somas_prefixos;
	hipMalloc(&device_arr, bytes);
	hipMalloc(&device_somas_prefixos, bytes);
	
	// Copia da maquina hospedeira (host) para o dispositivo (device)
	hipMemcpy(device_arr, host_arr.data(), bytes, hipMemcpyHostToDevice);
	
	// Tamanho do bloco em número de threads
	const int TAMANHO_BLOCO = 8;

	// Tamanho do grid em número de bloco
    // (tamanho do array / número de threads por bloco)
	int TAMANHO_GRID = N / TAMANHO_BLOCO;

	// Chamadas para o kernel
	reducaoSoma<<<TAMANHO_GRID, TAMANHO_BLOCO>>>(device_arr, device_somas_prefixos);

	// Copia do dispositivo (device) para a máquina hospedeira (host)
	hipMemcpy(host_somas_prefixos.data(), device_somas_prefixos, bytes, hipMemcpyDeviceToHost);

	// Confere resultado
	// ## VERIFICAÇÃO ##
	// Implemente aqui a verificação do resultado do algoritmo paralelo
	assert(false);

	cout << "REDUÇÃO OCORREU COM SUCESSO.\n";

	return 0;
}

// ## SOMA DE PREFIXOS EM CPU ##
// Implemente aqui a soma de prefixos sequencial
void somaPrefixos(int *arr, int *somas, int tamanho) {}